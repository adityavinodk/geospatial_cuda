#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <sstream>
#include <vector>
using namespace std;

#define mp make_pair
#define fi first
#define se second

struct Point {
	int x, y;

	Point(int xc, int yc) : x(xc), y(yc) {}
};

struct Grid {
	Grid bottom_left, bottom_right, top_left, top_right;
	Point *points;

	// Initialize the corresponding Point values
	Grid(Grid bl, Grid br, Grid tl, Grid tr, Point *ps)
		: bottom_left(bl),
		  bottom_right(br),
		  top_left(tl),
		  top_right(tr),
		  points(ps) {}
};

__global__ void categorize_points(Point *d_points, int *d_categories,
								  int *grid_counts, int count, int range,
								  int middle_x, int middle_y) {
	// subgrid_counts declared outside kernel, Dynamic Shared Memory
	// Accessed using extern
	extern __shared__ int subgrid_counts[];

	int start = ((blockIdx.x * blockDim.x) + threadIdx.x) * range;

	// Initialize the subgrid counts to 0
	if (threadIdx.x == 0) {
		subgrid_counts[0] = 0;
		subgrid_counts[1] = 0;
		subgrid_counts[2] = 0;
		subgrid_counts[3] = 0;
	}
	__syncthreads();

	int first = 0, second = 0, third = 0, fourth = 0;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
			// bottom left; if the point lies in bottom left, increment
			if (d_points[i].x <= middle_x and d_points[i].y <= middle_y) {
				d_categories[i] = 1;
				first++;
			}
			// bottom right; if point lies in bottom right, increment
			else if (d_points[i].x > middle_x and d_points[i].y <= middle_y) {
				d_categories[i] = 2;
				second++;
			}
			// top left; if point lies in top left, increment
			else if (d_points[i].x <= middle_x and d_points[i].y > middle_y) {
				d_categories[i] = 3;
				third++;
			}
			// top right; if point lies in top right, increment
			else if (d_points[i].x > middle_x and d_points[i].y > middle_y) {
				d_categories[i] = 4;
				fourth++;
			}
		}
	}

	// CUDA built in function to perform atomic addition at given location
	// Location : first variable
	// Store the counts of points in their respective subgrid
	atomicAdd(&subgrid_counts[0], first);
	atomicAdd(&subgrid_counts[1], second);
	atomicAdd(&subgrid_counts[2], third);
	atomicAdd(&subgrid_counts[3], fourth);
	__syncthreads();

	// Add the values of subgrid_counts to grid_counts
	if (threadIdx.x == 0) {
		atomicAdd(&grid_counts[0], subgrid_counts[0]);
		atomicAdd(&grid_counts[1], subgrid_counts[1]);
		atomicAdd(&grid_counts[2], subgrid_counts[2]);
		atomicAdd(&grid_counts[3], subgrid_counts[3]);
	}
}

__global__ void organize_points(Point *d_points, int *d_categories, Point *bl,
								Point *br, Point *tl, Point *tr, int count,
								int range) {
	extern __shared__ int subgrid_index[];

	// Initialize subgrid pointer to 0
	// Used to index the point arrays for each subgrid
	if (threadIdx.x == 0) {
		subgrid_index[0] = 0;
		subgrid_index[1] = 0;
		subgrid_index[2] = 0;
		subgrid_index[3] = 0;
	}
	__syncthreads();

	for (int i = threadIdx.x; i < threadIdx.x + range; i++) {
		if (i < count) {
			// Point array will store the respective points in a contiguous
			// fashion increment subgrid index according to the category
			if (d_categories[i] == 1) {
				bl[subgrid_index[0]] = d_points[i];
				atomicAdd(&subgrid_index[0], 1);
			}
			if (d_categories[i] == 2) {
				br[subgrid_index[1]] = d_points[i];
				atomicAdd(&subgrid_index[1], 1);
			}
			if (d_categories[i] == 3) {
				tl[subgrid_index[2]] = d_points[i];
				atomicAdd(&subgrid_index[2], 1);
			}
			if (d_categories[i] == 4) {
				tr[subgrid_index[3]] = d_points[i];
				atomicAdd(&subgrid_index[3], 1);
			}
		}
	}
}

void quadtree_grid(vector<Point> points, int count,
				   pair<int, int> bottom_left_corner,
				   pair<int, int> top_right_corner) {
	// Array of points for the geospatial data
	Point *d_points;

	// array to store the category of points (size = count) and the count of
	// points in each grid (size = 4)
	int *d_categories, *d_grid_counts;

	// Declare vectors to store the final values.
	vector<int> h_categories(count);
	vector<int> h_grid_counts(4);

	// Allocate memory to the pointers
	hipMalloc(&d_points, count * sizeof(Point));
	hipMalloc(&d_categories, count * sizeof(int));
	hipMalloc(&d_grid_counts, 4 * sizeof(int));

	// Copy the point data into device
	hipMemcpy(d_points, points.data(), count * sizeof(Point),
			   hipMemcpyHostToDevice);

	// Set the number of blocks and threads per block
	int range, num_blocks = 16, threads_per_block = 256;

	// Calculate the work done by each thread
	if (count < num_blocks * threads_per_block)
		range = 1;
	else if (count % (num_blocks * threads_per_block) == 0)
		range = count / (threads_per_block * num_blocks);
	else {
		float value =
			static_cast<float>(count) / (num_blocks * threads_per_block);
		range = std::ceil(value);
	}
	printf("Categorize in GPU: %d blocks of %d threads each with range=%d\n",
		   num_blocks, threads_per_block, range);

	dim3 grid(num_blocks, 1, 1);
	dim3 block(threads_per_block, 1, 1);

	// KERNEL Function to categorize points into 4 subgrids
	int middle_x = (top_right_corner.fi - bottom_left_corner.fi) / 2,
		middle_y = (top_right_corner.se - bottom_left_corner.se) / 2;
	categorize_points<<<grid, block, 4 * sizeof(int)>>>(
		d_points, d_categories, d_grid_counts, count, range, middle_x,
		middle_y);

	// Get back the data from device to host
	hipMemcpy(h_categories.data(), d_categories, count * sizeof(int),
			   hipMemcpyDeviceToHost);
	hipMemcpy(h_grid_counts.data(), d_grid_counts, 4 * sizeof(int),
			   hipMemcpyDeviceToHost);

	int total = 0;
	printf("Point counts per sub grid - \n");
	for (int i = 0; i < 4; i++) {
		printf("sub grid %d - %d\n", i + 1, h_grid_counts[i]);
		total += h_grid_counts[i];
	}
	printf("Total Count - %d\n", count);
	if (total == count) {
		printf("Sum of sub grid counts matches total point count\n\n");
	}

	// Declare arrays for each section of the grid and allocate memory depending
	// on the number of points found
	Point *bottom_left, *bottom_right, *top_left, *top_right;
	hipMalloc(&bottom_left, h_grid_counts[0] * sizeof(Point));
	hipMalloc(&bottom_right, h_grid_counts[1] * sizeof(Point));
	hipMalloc(&top_left, h_grid_counts[2] * sizeof(Point));
	hipMalloc(&top_right, h_grid_counts[3] * sizeof(Point));

	dim3 grid2(1, 1, 1);
	dim3 block2(threads_per_block, 1, 1);

	// KERNEL Function to assign the points to its respective array
	float value = static_cast<float>(count) / threads_per_block;
	range = std::ceil(value);
	printf("Organize in GPU: 1 block of %d threads each with range=%d\n",
		   threads_per_block, range);
	organize_points<<<grid2, block2, 4 * sizeof(int)>>>(
		d_points, d_categories, bottom_left, bottom_right, top_left, top_right,
		count, count / threads_per_block);

	// Declare the final array in which we store the sorted points according to
	// the location in the grid
	Point *bl, *br, *tl, *tr;
	bl = (Point *)malloc(h_grid_counts[0] * sizeof(Point));
	br = (Point *)malloc(h_grid_counts[1] * sizeof(Point));
	tl = (Point *)malloc(h_grid_counts[2] * sizeof(Point));
	tr = (Point *)malloc(h_grid_counts[3] * sizeof(Point));

	// Shift the data from device to host
	hipMemcpy(bl, bottom_left, h_grid_counts[0] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(br, bottom_right, h_grid_counts[1] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tl, top_left, h_grid_counts[2] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tr, top_right, h_grid_counts[3] * sizeof(Point),
			   hipMemcpyDeviceToHost);

	printf("Sample Point in bottom left - %d %d\n", bl[0].x, bl[0].y);
	printf("Sample Point in bottom right - %d %d\n", br[0].x, br[0].y);
	printf("Sample Point in top left - %d %d\n", tl[0].x, tl[0].y);
	printf("Sample Point in top right - %d %d\n", tr[0].x, tr[0].y);

	// TODO:
	// Recursively call the quadtree grid function on each of the 4 sub grids -
	// bl, br, tl, tr and store in Grid struct

	// Free data
	hipFree(d_points);
	hipFree(d_categories);
	hipFree(d_grid_counts);
	hipFree(bottom_left);
	hipFree(bottom_right);
	hipFree(top_left);
	hipFree(top_right);
}

int main() {
	string filename = "points.txt";
	vector<Point> points;
	int point_count = 0;

	ifstream file(filename);
	if (!file) {
		cerr << "Error: Could not open the file " << filename << endl;
		return 1;
	}

	string line;
	int x, y;

	while (getline(file, line)) {
		istringstream iss(line);
		if (iss >> x >> y) {
			Point p = Point(x, y);
			points.emplace_back(p);
			point_count++;
		} else {
			cerr << "Warning: Skipping malformed line: " << line << endl;
		}
	}

	file.close();

	quadtree_grid(points, point_count, mp(0, 0), mp(1000, 1000));

	return 0;
}

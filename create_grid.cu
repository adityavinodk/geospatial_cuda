#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <sstream>
#include <vector>
using namespace std;

struct Point {
	int x, y;

	Point(int xc, int yc) : x(xc), y(yc) {}
};

struct Grid {
	Point *bottom_left, *bottom_right, *top_left, *top_right;

	// Initialize the corresponding Point values
	Grid(Point *bl, Point *br, Point *tl, Point *tr)
		: bottom_left(bl), bottom_right(br), top_left(tl), top_right(tr) {}
};

// NODE:
struct Node {
	int x_min, x_max, y_min, y_max;
	int start_idx;
	int end_idx;
	Node *bl, *br, *ul, *ur;
};

__global__ void categorize_points(Point *d_points, int *d_categories,
								  int *grid_counts, int count, int range,
								  int middle) {
	// subgrid_counts declared outside kernel, Dynamic Shared Memory
	// Accessed using extern
	extern __shared__ int subgrid_counts[];

	int start = ((blockIdx.x * blockDim.x) + threadIdx.x) * range;

	// Initialize the subgrid counts to 0
	if (threadIdx.x == 0) {
		subgrid_counts[0] = 0;
		subgrid_counts[1] = 0;
		subgrid_counts[2] = 0;
		subgrid_counts[3] = 0;
	}
	__syncthreads();

	int first = 0, second = 0, third = 0, fourth = 0;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
			// bottom left; if the point lies in bottom left, increment
			if (d_points[i].x <= middle and d_points[i].y <= middle) {
				d_categories[i] = 1;
				first++;
			}
			// bottom right; if point lies in bottom right, increment
			else if (d_points[i].x > middle and d_points[i].y <= middle) {
				d_categories[i] = 2;
				second++;
			}
			// top left; if point lies in top left, increment
			else if (d_points[i].x <= middle and d_points[i].y > middle) {
				d_categories[i] = 3;
				third++;
			}
			// top right; if point lies in top right, increment
			else if (d_points[i].x > middle and d_points[i].y > middle) {
				d_categories[i] = 4;
				fourth++;
			}
		}
	}

	// CUDA built in function to perform atomic addition at given location
	// Location : first variable
	// Store the counts of points in their respective subgrid
	atomicAdd(&subgrid_counts[0], first);
	atomicAdd(&subgrid_counts[1], second);
	atomicAdd(&subgrid_counts[2], third);
	atomicAdd(&subgrid_counts[3], fourth);
	__syncthreads();

	// Add the values of subgrid_counts to grid_counts
	if (threadIdx.x == 0) {
		atomicAdd(&grid_counts[0], subgrid_counts[0]);
		atomicAdd(&grid_counts[1], subgrid_counts[1]);
		atomicAdd(&grid_counts[2], subgrid_counts[2]);
		atomicAdd(&grid_counts[3], subgrid_counts[3]);
	}
}

__global__ void organize_points(Point *d_points, int *d_categories, Point *bl,
								Point *br, Point *tl, Point *tr, int count,
								int range) {
	extern __shared__ int subgrid_index[];

	// Initialize subgrid pointer to 0
	// Used to index the point arrays for each subgrid
	if (threadIdx.x == 0) {
		subgrid_index[0] = 0;
		subgrid_index[1] = 0;
		subgrid_index[2] = 0;
		subgrid_index[3] = 0;
	}
	__syncthreads();

	for (int i = threadIdx.x; i < threadIdx.x + range; i++) {
		if (i < count) {
			// Point array will store the respective points in a contiguous
			// fashion increment subgrid index according to the category
			if (d_categories[i] == 1) {
				bl[subgrid_index[0]] = d_points[i];
				atomicAdd(&subgrid_index[0], 1);
			}
			if (d_categories[i] == 2) {
				br[subgrid_index[1]] = d_points[i];
				atomicAdd(&subgrid_index[1], 1);
			}
			if (d_categories[i] == 3) {
				tl[subgrid_index[2]] = d_points[i];
				atomicAdd(&subgrid_index[2], 1);
			}
			if (d_categories[i] == 4) {
				tr[subgrid_index[3]] = d_points[i];
				atomicAdd(&subgrid_index[3], 1);
			}
		}
	}
}

// ---------- UTIL FUNCTIONS ---------------------

__global__ void assignBuckets(Point* points, int* bucketCounts, int* sortedIndices, 
                              float xMin, float xRange, float yMin, float yRange, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        Point p = points[idx];
        int xIndex = 2 * (p.x - xMin) / xRange;
        int yIndex = 2 * (p.y - yMin) / yRange;
        int bucketIndex = 2 * yIndex + xIndex;
        
        // Atomic add to bucket counts for each bucket
        atomicAdd(&bucketCounts[bucketIndex], 1);

        // Save the bucket index for sorting later
        sortedIndices[idx] = bucketIndex;
    }
}

__global__ void reorderPoints(Point* points, Point* sortedPoints, int* sortedIndices, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        int bucket = sortedIndices[idx];
        int position = atomicAdd(&bucketPositions[bucket], 1);  // Position within the sorted bucket
        sortedPoints[position] = points[idx];
    }
}

__global__ void findMinMax(Point* points, float* xMin, float* xMax, float* yMin, float* yMax, int n) {
    //dynamic shared memory: holding min and max of the block, used for further reduction
	extern __shared__ float sharedMinMax[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize local variables for min and max
    float localXMin = (idx < n) ? x[idx] : FLT_MAX;
    float localXMax = (idx < n) ? x[idx] : -FLT_MAX;
    float localYMin = (idx < n) ? y[idx] : FLT_MAX;
    float localYMax = (idx < n) ? y[idx] : -FLT_MAX;

	// Store local mins and maxes in shared memory
    sharedMinMax[tid] = localXMin;                  
    sharedMinMax[blockDim.x + tid] = localXMax;
    sharedMinMax[2 * blockDim.x + tid] = localYMin;
    sharedMinMax[3 * blockDim.x + tid] = localYMax;

    __syncthreads();

    // Performing reduction in shared memory to find min and max for each block
    for (int s = blockDim.x / 2; s > 0; s = s/2) {
        if (tid < s) {
            sharedMinMax[tid] = fminf(sharedMinMax[tid], sharedMinMax[tid + s]);            // Reduce x min
            sharedMinMax[blockDim.x + tid] = fmaxf(sharedMinMax[blockDim.x + tid], sharedMinMax[blockDim.x + tid + s]); // Reduce x max
            sharedMinMax[2 * blockDim.x + tid] = fminf(sharedMinMax[2 * blockDim.x + tid], sharedMinMax[2 * blockDim.x + tid + s]); // Reduce y min
            sharedMinMax[3 * blockDim.x + tid] = fmaxf(sharedMinMax[3 * blockDim.x + tid], sharedMinMax[3 * blockDim.x + tid + s]); // Reduce y max
        }
        __syncthreads();
    }

    // Once we reach this point we will have each block's min and max so we write the result for min/max of all blocks to global memory
    if (tid == 0) {
        atomicMin(xMin, sharedMinMax[0]);
        atomicMax(xMax, sharedMinMax[blockDim.x]);
        atomicMin(yMin, sharedMinMax[2 * blockDim.x]);
        atomicMax(yMax, sharedMinMax[3 * blockDim.x]);
    }
}

// -------------------------------


void quadtree_grid(vector<Point> points, int count, int dimension) {
	// Array of points for the geospatial data
	Point *d_points;

	// array to store the category of points (size = count) and the count of
	// points in each grid (size = 4)
	int *d_categories, *d_grid_counts;

	// Declare vectors to store the final values.
	vector<int> h_categories(count);
	vector<int> h_grid_counts(4);

	// Allocate memory to the pointers
	hipMalloc(&d_points, count * sizeof(Point));
	hipMalloc(&d_categories, count * sizeof(int));
	hipMalloc(&d_grid_counts, 4 * sizeof(int));

	// Copy the point data into device
	hipMemcpy(d_points, points.data(), count * sizeof(Point),
			   hipMemcpyHostToDevice);

	// Set the number of blocks and threads per block
	int range, num_blocks = 16, threads_per_block = 256;

	// Calculate the work done by each thread
	if (count < num_blocks * threads_per_block)
		range = 1;
	else if (count % (num_blocks * threads_per_block) == 0)
		range = count / (threads_per_block * num_blocks);
	else {
		float value =
			static_cast<float>(count) / (num_blocks * threads_per_block);
		range = std::ceil(value);
	}
	printf("GPU: %d blocks of %d threads each with range=%d\n", num_blocks,
		   threads_per_block, range);

	dim3 grid(num_blocks, 1, 1);
	dim3 block(threads_per_block, 1, 1);

	// KERNEL Function to categorize points into 4 subgrids
	categorize_points<<<grid, block, 4 * sizeof(int)>>>(
		d_points, d_categories, d_grid_counts, count, range, dimension / 2);

	// Get back the data from device to host
	hipMemcpy(h_categories.data(), d_categories, count * sizeof(int),
			   hipMemcpyDeviceToHost);
	hipMemcpy(h_grid_counts.data(), d_grid_counts, 4 * sizeof(int),
			   hipMemcpyDeviceToHost);

	// for(int i = 0; i<1000; i++){
	// printf("x = %d, y = %d, category = %d\n", points[i].x, points[i].y,
	// h_categories[i]);
	//}
	// int total = 0;
	// for(int i = 0; i<4; i++){
	// printf("sub grid %d - %d\n", i+1, h_grid_counts[i]);
	// total += h_grid_counts[i];
	//}
	// printf("Total Count - %d\n", count);
	// if(total == count){
	// printf("Matches\n");
	//}

	// Declare arrays for each section of the grid and allocate memory depending
	// on the number of points found
	Point *bottom_left, *bottom_right, *top_left, *top_right;
	hipMalloc(&bottom_left, h_grid_counts[0] * sizeof(Point));
	hipMalloc(&bottom_right, h_grid_counts[1] * sizeof(Point));
	hipMalloc(&top_left, h_grid_counts[2] * sizeof(Point));
	hipMalloc(&top_right, h_grid_counts[3] * sizeof(Point));

	dim3 grid2(1, 1, 1);
	dim3 block2(threads_per_block, 1, 1);

	// KERNEL Function to assign the points to its respective array
	organize_points<<<grid2, block2, 4 * sizeof(int)>>>(
		d_points, d_categories, bottom_left, bottom_right, top_left, top_right,
		count, count / threads_per_block);

	// Declare the final array in which we store the sorted points according to
	// the location in the grid
	Point *bl, *br, *tl, *tr;
	bl = (Point *)malloc(h_grid_counts[0] * sizeof(Point));
	br = (Point *)malloc(h_grid_counts[1] * sizeof(Point));
	tl = (Point *)malloc(h_grid_counts[2] * sizeof(Point));
	tr = (Point *)malloc(h_grid_counts[3] * sizeof(Point));

	// Shift the data from device to host
	hipMemcpy(bl, bottom_left, h_grid_counts[0] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(br, bottom_right, h_grid_counts[1] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tl, top_left, h_grid_counts[2] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tr, top_right, h_grid_counts[3] * sizeof(Point),
			   hipMemcpyDeviceToHost);

	printf("Point in bottom left - %d %d\n", bl[0].x, bl[0].y);
	printf("Point in bottom right - %d %d\n", br[0].x, br[0].y);
	printf("Point in top left - %d %d\n", tl[0].x, tl[0].y);
	printf("Point in top right - %d %d\n", tr[0].x, tr[0].y);

	// Free data
	hipFree(d_points);
	hipFree(d_categories);
	hipFree(d_grid_counts);
	hipFree(bottom_left);
	hipFree(bottom_right);
	hipFree(top_left);
	hipFree(top_right);
}

int main() {
	string filename = "points.txt";
	vector<Point> points;
	int point_count = 0;

	ifstream file(filename);
	if (!file) {
		cerr << "Error: Could not open the file " << filename << endl;
		return 1;
	}

	string line;
	int x, y;

	while (getline(file, line)) {
		istringstream iss(line);
		if (iss >> x >> y) {
			Point p = Point(x, y);
			points.emplace_back(p);
			point_count++;
		} else {
			cerr << "Warning: Skipping malformed line: " << line << endl;
		}
	}

	file.close();

	quadtree_grid(points, point_count, 1000);

	return 0;
}

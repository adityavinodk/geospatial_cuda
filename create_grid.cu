#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <sstream>
#include <vector>
using namespace std;

struct Point {
	int x, y;

	Point(int xc, int yc) : x(xc), y(yc) {}
};

struct Grid {
    Point *bottom_left, *bottom_right, *top_left, *top_right;

    // Initialize the corresponding Point values
    Grid(Point* bl, Point* br, Point* tl, Point* tr)
                : bottom_left(bl), bottom_right(br), top_left(tl), top_right(tr) {}
};

__global__ void categorize_points(Point *d_points, int *d_categories, int *grid_counts, int count,
								  int range, int middle) {

    // subgrid_counts declared outside kernel, Dynamic Shared Memory
    // Accessed using extern
    extern __shared__ int subgrid_counts[];

	int start = ((blockIdx.x * blockDim.x) + threadIdx.x) * range;

    // Initialize the subgrid counts to 0
    if (threadIdx.x == 0) {
        subgrid_counts[0] = 0;
        subgrid_counts[1] = 0;
        subgrid_counts[2] = 0;
        subgrid_counts[3] = 0;
    }
    __syncthreads();

    int first = 0, second = 0, third = 0, fourth = 0;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
            // bottom left; if the point lies in bottom left, increment
            if (d_points[i].x <= middle and d_points[i].y <= middle){
				d_categories[i] = 1;
                first++;
            }
            // bottom right; if point lies in bottom right, increment
            else if (d_points[i].x > middle and d_points[i].y <= middle){
				d_categories[i] = 2;
                second++;
            }
            // top left; if point lies in top left, increment
            else if (d_points[i].x <= middle and d_points[i].y > middle){
				d_categories[i] = 3;
                third++;
            }
            // top right; if point lies in top right, increment
            else if (d_points[i].x > middle and d_points[i].y > middle){
				d_categories[i] = 4;
                fourth++;
            }
		}
	}

    // CUDA built in function to perform atomic addition at given location
    // Location : first variable
    // Store the counts of points in their respective subgrid
    atomicAdd(&subgrid_counts[0], first);
    atomicAdd(&subgrid_counts[1], second);
    atomicAdd(&subgrid_counts[2], third);
    atomicAdd(&subgrid_counts[3], fourth);
    __syncthreads();

    // Add the values of subgrid_counts to grid_counts
    if (threadIdx.x == 0) {
        atomicAdd(&grid_counts[0], subgrid_counts[0]);
        atomicAdd(&grid_counts[1], subgrid_counts[1]);
        atomicAdd(&grid_counts[2], subgrid_counts[2]);
        atomicAdd(&grid_counts[3], subgrid_counts[3]);
    }
}

__global__ void organize_points(Point *d_points, int *d_categories, Point* bl, Point* br, Point* tl, Point* tr, int count, int range) {
    extern __shared__ int subgrid_index[];

    // Initialize subgrid pointer to 0
    // Used to index the point arrays for each subgrid
    if (threadIdx.x == 0) {
        subgrid_index[0] = 0;
        subgrid_index[1] = 0;
        subgrid_index[2] = 0;
        subgrid_index[3] = 0;
    }
    __syncthreads();
    
    
	for (int i = threadIdx.x; i < threadIdx.x + range; i++) {
		if (i < count) {
            // Point array will store the respective points in a contiguous fashion
            // increment subgrid index according to the category
            if(d_categories[i] == 1) {
                bl[subgrid_index[0]] = d_points[i];
                atomicAdd(&subgrid_index[0], 1);
            }
            if(d_categories[i] == 2) {
                br[subgrid_index[1]] = d_points[i];
                atomicAdd(&subgrid_index[1], 1);
            }
            if(d_categories[i] == 3) {
                tl[subgrid_index[2]] = d_points[i];
                atomicAdd(&subgrid_index[2], 1);
            }
            if(d_categories[i] == 4) {
                tr[subgrid_index[3]] = d_points[i];
                atomicAdd(&subgrid_index[3], 1);
            }
		}
	}
}

void quadtree_grid(vector<Point> points, int count, int dimension) {
    // Array of points for the geospatial data
	Point *d_points;

    // array to store the category of points (size = count) and the count of points in each grid (size = 4)
	int *d_categories, *d_grid_counts;

    // Declare vectors to store the final values.
    vector<int> h_categories(count);
    vector<int> h_grid_counts(4);

    // Allocate memory to the pointers
	hipMalloc(&d_points, count * sizeof(Point));
	hipMalloc(&d_categories, count * sizeof(int));
	hipMalloc(&d_grid_counts, 4 * sizeof(int));

    // Copy the point data into device
	hipMemcpy(d_points, points.data(), count * sizeof(Point),
			   hipMemcpyHostToDevice);

    // Set the number of blocks and threads per block
	int range, num_blocks = 16, threads_per_block = 256;

    // Calculate the work done by each thread
	if (count < num_blocks * threads_per_block)
		range = 1;
	else if (count % (num_blocks * threads_per_block) == 0)
		range = count / (threads_per_block * num_blocks);
	else {
		float value = static_cast<float>(count) / (num_blocks * threads_per_block);
		range = std::ceil(value);
	}
	printf("GPU: %d blocks of %d threads each with range=%d\n", num_blocks,
		   threads_per_block, range);

	dim3 grid(num_blocks, 1, 1);
	dim3 block(threads_per_block, 1, 1);

    // KERNEL Function to categorize points into 4 subgrids
	categorize_points<<<grid, block, 4 * sizeof(int)>>>(d_points, d_categories, d_grid_counts, count, range,
									   dimension / 2);

    // Get back the data from device to host
	hipMemcpy(h_categories.data(), d_categories, count * sizeof(int),
			   hipMemcpyDeviceToHost);
	hipMemcpy(h_grid_counts.data(), d_grid_counts, 4 * sizeof(int),
			   hipMemcpyDeviceToHost);


    //for(int i = 0; i<1000; i++){
        //printf("x = %d, y = %d, category = %d\n", points[i].x, points[i].y, h_categories[i]);
    //}
    //int total = 0;
    //for(int i = 0; i<4; i++){
        //printf("sub grid %d - %d\n", i+1, h_grid_counts[i]);
        //total += h_grid_counts[i];
    //}
    //printf("Total Count - %d\n", count);
    //if(total == count){
        //printf("Matches\n");
    //}

    // Declare arrays for each section of the grid and allocate memory depending on the number of points found
	Point *bottom_left, *bottom_right, *top_left, *top_right;
	hipMalloc(&bottom_left, h_grid_counts[0] * sizeof(Point));
	hipMalloc(&bottom_right, h_grid_counts[1] * sizeof(Point));
	hipMalloc(&top_left, h_grid_counts[2] * sizeof(Point));
	hipMalloc(&top_right, h_grid_counts[3] * sizeof(Point));

	dim3 grid2(1, 1, 1);
	dim3 block2(threads_per_block, 1, 1);

    // KERNEL Function to assign the points to its respective array
	organize_points<<<grid2, block2, 4 * sizeof(int)>>>(d_points, d_categories, bottom_left, bottom_right, top_left, top_right, count, count / threads_per_block);

    // Declare the final array in which we store the sorted points according to the location in the grid
    Point *bl, *br, *tl, *tr;
    bl = (Point*)malloc(h_grid_counts[0] * sizeof(Point));
    br = (Point*)malloc(h_grid_counts[1] * sizeof(Point));
    tl = (Point*)malloc(h_grid_counts[2] * sizeof(Point));
    tr = (Point*)malloc(h_grid_counts[3] * sizeof(Point));

    // Shift the data from device to host
	hipMemcpy(bl, bottom_left, h_grid_counts[0] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(br, bottom_right, h_grid_counts[1] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tl, top_left, h_grid_counts[2] * sizeof(Point),
			   hipMemcpyDeviceToHost);
	hipMemcpy(tr, top_right, h_grid_counts[3] * sizeof(Point),
			   hipMemcpyDeviceToHost);

    printf("Point in bottom left - %d %d\n", bl[0].x, bl[0].y);
    printf("Point in bottom right - %d %d\n", br[0].x, br[0].y);
    printf("Point in top left - %d %d\n", tl[0].x, tl[0].y);
    printf("Point in top right - %d %d\n", tr[0].x, tr[0].y);

    // Free data 
    hipFree(d_points);
    hipFree(d_categories);
    hipFree(d_grid_counts);
    hipFree(bottom_left);
    hipFree(bottom_right);
    hipFree(top_left);
    hipFree(top_right);
}

int main() {
	string filename = "points.txt";
	vector<Point> points;
	int point_count = 0;

	ifstream file(filename);
	if (!file) {
		cerr << "Error: Could not open the file " << filename << endl;
		return 1;
	}

	string line;
	int x, y;

	while (getline(file, line)) {
		istringstream iss(line);
		if (iss >> x >> y) {
			Point p = Point(x, y);
			points.emplace_back(p);
			point_count++;
		} else {
			cerr << "Warning: Skipping malformed line: " << line << endl;
		}
	}

	file.close();

	quadtree_grid(points, point_count, 1000);

	return 0;
}

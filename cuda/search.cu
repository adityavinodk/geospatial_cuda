#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

#include <cmath>
#include <fstream>
#include <sstream>
#include <vector>
#include <unordered_map>

using namespace std;

int search_quadrant(Point target_point, const vector<QuadrantBoundary> &boundaries)
{
    QuadrantBoundary *d_boundaries;
    hipMalloc(&d_boundaries, boundaries.size() * sizeof(QuadrantBoundary));
    hipMemcpy(d_boundaries, boundaries.data(), boundaries.size() * sizeof(QuadrantBoundary), hipMemcpyHostToDevice);

    Point *d_target_point;
    hipMalloc(&d_target_point, sizeof(Point));
    hipMemcpy(d_target_point, &target_point, sizeof(Point), hipMemcpyHostToDevice);

    int *d_result;
    hipMalloc(&d_result, sizeof(int));

    int init_value = -1;
    hipMemcpy(d_result, &init_value, sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = 16;
    quadrant_search<<<num_blocks, block_size>>>(d_target_point, d_boundaries, boundaries.size(), d_result);

    int result;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_boundaries);
    hipFree(d_target_point);
    hipFree(d_result);

    return (result == -1) ? -1 : result; // Return -1 if point not found in any quadrant
}

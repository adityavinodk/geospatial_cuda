#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <kernels.h>

using namespace std;

__global__ void categorize_points(Point *d_points, int *d_categories,
								  int *grid_counts, int count, int range,
								  float middle_x, float middle_y)
{
	// subgrid_counts declared outside kernel, Dynamic Shared Memory
	// Accessed using extern
	extern __shared__ int subgrid_counts[];

	int start = ((blockIdx.x * blockDim.x) + threadIdx.x) * range;

	// Initialize the subgrid counts to 0
	if (threadIdx.x == 0) {
		subgrid_counts[0] = 0;
		subgrid_counts[1] = 0;
		subgrid_counts[2] = 0;
		subgrid_counts[3] = 0;
	}
	__syncthreads();

	int first = 0, second = 0, third = 0, fourth = 0;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
			// bottom left; if the point lies in bottom left, increment
			if (d_points[i].x <= middle_x and d_points[i].y <= middle_y) {
				d_categories[i] = 0;
				first++;
			}
			// bottom right; if point lies in bottom right, increment
			else if (d_points[i].x > middle_x and d_points[i].y <= middle_y) {
				d_categories[i] = 1;
				second++;
			}
			// top left; if point lies in top left, increment
			else if (d_points[i].x <= middle_x and d_points[i].y > middle_y) {
				d_categories[i] = 2;
				third++;
			}
			// top right; if point lies in top right, increment
			else if (d_points[i].x > middle_x and d_points[i].y > middle_y) {
				d_categories[i] = 3;
				fourth++;
			}
		}
	}

	// CUDA built in function to perform atomic addition at given location
	// Location : first variable
	// Store the counts of points in their respective subgrid
	atomicAdd(&subgrid_counts[0], first);
	atomicAdd(&subgrid_counts[1], second);
	atomicAdd(&subgrid_counts[2], third);
	atomicAdd(&subgrid_counts[3], fourth);
	__syncthreads();

	// Add the values of subgrid_counts to grid_counts
	if (threadIdx.x == 0) {
		atomicAdd(&grid_counts[0], subgrid_counts[0]);
		atomicAdd(&grid_counts[1], subgrid_counts[1]);
		atomicAdd(&grid_counts[2], subgrid_counts[2]);
		atomicAdd(&grid_counts[3], subgrid_counts[3]);
	}
}

__global__ void organize_points(Point *d_points, int *d_categories, Point *bl,
								Point *br, Point *tl, Point *tr, int count,
								int range) {
	extern __shared__ int subgrid_index[];

	// Initialize subgrid pointer to 0
	// Used to index the point arrays for each subgrid
	if (threadIdx.x == 0) {
		subgrid_index[0] = 0;
		subgrid_index[1] = 0;
		subgrid_index[2] = 0;
		subgrid_index[3] = 0;
	}
	__syncthreads();

	int start = threadIdx.x * range;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
			// Point array will store the respective points in a contiguous
			// fashion increment subgrid index according to the category
			unsigned int category_index =
				atomicAdd(&subgrid_index[d_categories[i]], 1);
			if (d_categories[i] == 0) {
				bl[category_index] = d_points[i];
			}
			if (d_categories[i] == 1) {
				br[category_index] = d_points[i];
			}
			if (d_categories[i] == 2) {
				tl[category_index] = d_points[i];
			}
			if (d_categories[i] == 3) {
				tr[category_index] = d_points[i];
			}
		}
	}
}

//Quandrant Search to find the level of the quadrant where the point lies
__global__ void quadrant_search(Point *target_point, QuadrantBoundary *boundaries, int num_boundaries, int *result)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_boundaries)
	{
		QuadrantBoundary boundary = boundaries[idx];
		if (target_point->x >= boundary.bottom_left.first && target_point->x <= boundary.top_right.first &&
			target_point->y >= boundary.bottom_left.second && target_point->y <= boundary.top_right.second)
		{
			atomicMax(result, boundary.id);
		}
	}
}

// Validation Function
bool validateGrid(Grid* root_grid, pair<float, float>& TopRight, pair<float, float>& BottomLeft){
    if(root_grid == nullptr)
        return true;

    // If we have reached the bottom of the grid, we start validation
    if(root_grid -> points) {
        Point* point_array = root_grid -> points;
        float Top_x = TopRight.first;
        float Top_y = TopRight.second;

        float Bot_x = BottomLeft.first;
        float Bot_y = BottomLeft.second;

        float Mid_x = (Top_x + Bot_x) / 2;
        float Mid_y = (Top_y + Bot_y) / 2;

        int count = root_grid -> count;

        for(int i = 0; i < count; i ++){
            float point_x = point_array[i].x;
            float point_y = point_array[i].y;

            if(point_x < Bot_x || point_x > Top_x){
                printf("Validation Error! Point (%f, %f) is plced out of bounds. Grid dimension: [(%f, %f), (%f, %f)]\n", point_x, point_y, Bot_x, Bot_y, Top_x, Top_y);
                return false;
            }
            else if(point_y < Bot_y || point_y > Top_y){
                printf("Validation Error! Point (%f, %f) is plced out of bounds. Grid dimension: [(%f, %f), (%f, %f)]\n", point_x, point_y, Bot_x, Bot_y, Top_x, Top_y);
                return false;
            }
            else{
                continue;
            }
        }

        return true;
    }

    // Call Recursively for all 4 quadrants
    Grid* top_left_child     = nullptr;
    Grid* top_right_child    = nullptr;
    Grid* bottom_left_child  = nullptr;
    Grid* bottom_right_child = nullptr;

    top_left_child     = root_grid -> top_left;
    top_right_child    = root_grid -> top_right;
    bottom_left_child  = root_grid -> bottom_left;
    bottom_right_child = root_grid -> bottom_right;

    bool check_topLeft     = validateGrid(top_left_child, top_left_child->topRight, top_left_child->bottomLeft);
    bool check_topRight    = validateGrid(top_right_child, top_right_child->topRight, top_right_child->bottomLeft);
    bool check_bottomLeft  = validateGrid(bottom_left_child, bottom_left_child->topRight, bottom_left_child->bottomLeft);
    bool check_bottomRight = validateGrid(bottom_right_child, bottom_right_child->topRight, bottom_right_child->bottomLeft);

    return check_topLeft && check_topRight && check_bottomLeft && check_bottomRight;
}
